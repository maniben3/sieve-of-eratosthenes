
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

__global__ void init_primes_kernel(int *prime, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	while(index + offset < n){
		prime[index + offset] = index + offset + 1;

		offset += stride;
	}
}


__global__ void sieve_of_eratosthenes_kernel(int *prime, unsigned int n, unsigned int sqrRootN)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x + 2;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	while(index + offset < sqrRootN){
		unsigned int temp = index + offset;
		for(unsigned int i=temp*temp;i<n;i+=temp){
			prime[i-1] = 0;
		}

		offset += stride;
	}

}


int main()
{
	unsigned int N = 1*100*1024*1000;
	unsigned int M = (unsigned int)sqrt(N);
	int *h_primes;
	int *d_primes;


	// allocate memory
	h_primes = (int*)malloc(N*sizeof(int));
	hipMalloc((void**)&d_primes, N*sizeof(int));


	// timing on gpu
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);


	// call kernel
	dim3 gridSize = 32;
	dim3 blockSize = 32;
	init_primes_kernel<<< gridSize, blockSize >>>(d_primes, N);
	sieve_of_eratosthenes_kernel<<< gridSize, blockSize >>>(d_primes, N, M);


	// copy results back to host
	hipMemcpy(h_primes, d_primes, N*sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);
	std::cout<<"GPU took: "<<gpu_elapsed_time<<std::endl;

	// cpu version
	for(unsigned int i=0;i<N;i++){
		h_primes[i] = i+1;
	}
	clock_t cpu_start = clock();
	for(unsigned int i=0;i<M;i++){
		unsigned int start = (i+2)*(i+2);
		for(unsigned int j=start;j<N;j+=(i+2)){
			h_primes[j-1] = 0;
		}
	}
	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;
	std::cout<<"The cpu took: "<<cpu_elapsed_time<<" milli-seconds"<<std::endl;

	 for(unsigned int i=0;i<N;i++){
	 	if(h_primes[i] != 0){
	 		std::cout<<h_primes[i]<<"  ";
	 	}
	 }
	 std::cout<<""<<std::endl;


	// free memory
	free(h_primes);
	hipFree(d_primes);
}
